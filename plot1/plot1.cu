/**
 * @file Plot1.cu
 * @author Aniruddha Kulkarni (akulka24@ncsu.edu)
 * @brief
 *
 * Plot 1
 * C(x) = 4A(x)4– 2*A(x)2D(x)+ 8*A(x)3B(x) + 7*A(x)2B(x)2 + 5*A(x)B(x)3 + 2B(x)2 + 3*B(x)4 + 1
 *
 * To do:
 *
 * 1. Implement the base and optimized kernel
 * 2. Re-direct the std output to a txt file
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

/**
 * @brief
 *
 */
#define OPT 1

/**
 * @brief
 *
 * @param A
 * @param B
 * @param C
 * @param D
 * @param numElements
 * @return __global__
 */
__global__ void plot1(const float *A, const float *B, float *C, float *D, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

#if OPT == 1

    if (i < numElements)
    {
        float asq = A[i] * A[i];
        float bsq = B[i] * B[i];

        C[i] = asq * (4 * asq - 2 * D[i] + 8 * A[i] * B[i] + 7 * bsq) + bsq * (5 * A[i] * B[i] + 2 + 3 * bsq) + 1;
    }
#endif

#if OPT == 0

    if (i < numElements)
    {
        C[i] = 4 * A[i] * A[i] * A[i] * A[i] - 2 * A[i] * A[i] * D[i] + 8 * A[i] * A[i] * A[i] * B[i] + 7 * A[i] * A[i] * B[i] * B[i] + 5 * A[i] * B[i] * B[i] * B[i] + 2 * B[i] * B[i] + 3 * B[i] * B[i] * B[i] * B[i] + 1;
    }

#endif
}

int main(void)
{
    hipError_t err = hipSuccess;

    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Plot 1 Vector Addition calculation for %d elements]\n", numElements);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    float *h_D = (float *)malloc(size);

    if (h_A == NULL || h_B == NULL || h_C == NULL || h_D == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
        h_D[i] = rand() / (float)RAND_MAX;
    }

    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_D = NULL;
    err = hipMalloc((void **)&d_D, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector D (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_D, h_D, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector D from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    plot1<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, d_D, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < numElements; ++i)
    {

        if (fabs((h_C[i] - (4 * h_A[i] * h_A[i] * h_A[i] * h_A[i] - 2 * h_A[i] * h_A[i] * h_D[i] + 8 * h_A[i] * h_A[i] * h_A[i] * h_B[i] + 7 * h_A[i] * h_A[i] * h_B[i] * h_B[i] + 5 * h_A[i] * h_B[i] * h_B[i] * h_B[i] + 2 * h_B[i] * h_B[i] + 3 * h_B[i] * h_B[i] * h_B[i] * h_B[i] + 1)) / h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    err = hipFree(d_A);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_D);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector D (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);

    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}
